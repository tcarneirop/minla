#include "hip/hip_runtime.h"
#include <limits>
#include <iostream>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <chrono>
#include "../headers/grafo.h"
#include "../headers/full_perm.h"
#include "../headers/minla_node.h"
#include "../headers/minla_kernels.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void minla_gpu_node_explorer(int cutoff_depth, unsigned long long *tree_size, int *qtd_sols, int *best_sols,
    int* adj_list, Minla_node *pool, int pool_size,int upper_bound){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
     if(idx<pool_size){ //idx check

    //     unsigned flag = 0;
    //     unsigned bit_test = 0;
    //     int permutation[_MAX_]; //representa o ciclo
    //     int i, depth; //para dizer que 0-1 ja foi visitado e a busca comeca de 1, bote 2
     
    //     unsigned long long local_tree = 0ULL;
    //     int best_sol = upper_bound;
        
    //     int current_sol = 0;
    //     int num_sols = 0;
    //     int N = grafo->numNodes;
    //     int partial_cost = 0;
    //     int partial_sol = 0;    
    //     int stack[_MAX_];

    //     for (i = 0; i < N; ++i) { //
    //         permutation[i] = _EMPTY_;
    //     }

    //     depth = cutoff_depth;
    //     flag = pool[node_id].flag;
    //     partial_sol = pool[node_id].cost;

    //     for(i = 0; i<cutoff_depth;++i)
    //         permutation[i] = pool[node_id].permutation[i];
        
    //     while(true){ //search itself

    //         permutation[depth]++;
    //         bit_test = 0;
    //         bit_test |= (1<<permutation[depth]);

    //         if(permutation[depth] == N){ //all combinations for a given depth have been evaluated
    //             permutation[depth] = _EMPTY_;
    //         }
    //         else{

    //             if (!(flag & bit_test) ){ //is it valid?

    //                 partial_cost = grafo->ppartial_cost(permutation,depth+1);

    //                 if(partial_sol+partial_cost < best_sol){
                    
    //                     //cout<<" Leng: "<< depth+1<<" Partial cost: "<< current_sol<<" Cost test: "<<partial_sol+grafo->ppartial_cost(permutation,depth+1)<<"\n";
    //                     flag |= (1ULL<<permutation[depth]);
    //                     partial_sol += partial_cost; 
                        
    //                     stack[depth] = partial_cost;
                        
    //                     depth++;
    //                     ++local_tree;
                        
    //                     if (depth == N){ //a complete solution 
                            
    //                         ++num_sols;
    //                         best_sol = partial_sol;
    //                         //cout<<std::endl<<"Sol "<<num_sols<< " :"<<best_sol<<" "<<std::endl;
    //                         //for(int k = 1; k < N; k++){
    //                         //    std::cout   << " " << permutation[k];
    //                         //}

    //                     }//complete solution
    //                     else continue;
    //                 }//prune by value
    //                 else continue;
    //             }
    //             else continue;

    //         }//first else

    //         depth--; 
            
    //         //std::cout<<"\n alor";

    //         partial_sol-=stack[depth];
    //         flag &= ~(1ULL<<permutation[depth]);

    //         if(depth < cutoff_depth)
    //             break;
    //         //termination condition of the search

    //     }//end while -- end of the enumeration

        tree_size[idx] = 1;
        qtd_sols[idx]  = 1;
        best_sols[idx]  = 1;


    }//if idx   
 
}



void minla_call_multigpu_kernel(int gpu_id, int cutoff_depth, unsigned long long *tree_size,
    int *qtd_sols, Grafo *grafo, int upper_bound, int pool_size,  Minla_node *subsolutions_pool_h){
    
    hipSetDevice(gpu_id);
   // hipFuncSetCacheConfig(reinterpret_cast<const void*>(BP_queens_root_dfs),hipFuncCachePreferL1);
   
    unsigned long long *vector_of_tree_size_d;
    int *qtd_sols_d, *best_sol_d, *adj_list_d;
    
    Minla_node *subsolutions_pool_d;

    int local_qtd_sols = 0;
    unsigned long long local_tree_size = 0ULL;


    int num_blocks = ceil((double)pool_size/_MINLA_BLOCK_SIZE_);

    /////////////////////
    ////Host buffer
    ////////////////////
    int *qtd_sols_h =  (int *) malloc (sizeof(int)*pool_size);
    int *best_sol_h =  (int *) malloc (sizeof(int)*pool_size);
    int *adj_list_h =  (int *) malloc (sizeof(int)*grafo->numNodes*grafo->numNodes);
    unsigned long long *vector_of_tree_size_h = (unsigned long long *) malloc (sizeof(unsigned long long)*pool_size);


    /////////////////////
    ////GPU Buffer
    ////////////////////

    hipMalloc((void**) &vector_of_tree_size_d, pool_size*sizeof(unsigned long long));
    hipMalloc((void**) &adj_list_d, sizeof(int)*grafo->numNodes*grafo->numNodes);
    hipMalloc((void**) &qtd_sols_d,pool_size*sizeof(int));
    hipMalloc((void**) &best_sol_d, pool_size*sizeof(int));
    hipMalloc((void**) &subsolutions_pool_d,pool_size*sizeof(Minla_node));

    //memcopy 

    hipMemcpy(subsolutions_pool_d, subsolutions_pool_h, pool_size * sizeof(Minla_node), hipMemcpyHostToDevice);

    std::cout<<"### Regular BP-DFS search. ###\n";
    
    minla_gpu_node_explorer<<< num_blocks,_MINLA_BLOCK_SIZE_>>> (cutoff_depth, vector_of_tree_size_d, qtd_sols_d, best_sol_d,
        adj_list_d, subsolutions_pool_d, pool_size,upper_bound);   
 
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipPeekAtLastError() );
    std::cout<<"### End of the GPU search ###\n";

    hipMemcpy(vector_of_tree_size_h,vector_of_tree_size_d,pool_size*sizeof(unsigned long long),hipMemcpyDeviceToHost);
    hipMemcpy(qtd_sols_h,qtd_sols_d,pool_size*sizeof(int),hipMemcpyDeviceToHost);

    for(int i = 0; i<pool_size;++i){
        local_tree_size += vector_of_tree_size_h[i];
        local_qtd_sols  += qtd_sols_h[i];
    }

    std::cout<<"Local tree size: "<<local_tree_size<<"\n";
    std::cout<<"qtd_sols: "<<local_qtd_sols<<"\n";

    hipFree(vector_of_tree_size_d);
    hipFree(qtd_sols_d);
    hipFree(subsolutions_pool_d);
    //After that, Chapel reduces the values
}
