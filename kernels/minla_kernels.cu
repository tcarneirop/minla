#include "hip/hip_runtime.h"
#include <limits>
#include <iostream>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <chrono>
#include "../headers/grafo.h"
#include "../headers/fullperm.h"
#include "../headers/minla_node.h"
#include "../headers/minla_kernels.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void minla_call_multigpu_kernel(int gpu_id, int cutoff_depth, unsigned long long *local_tree_size,
    int *qtd_sols, Grafo *grafo, int upper_bound, int pool_size,  Minla_node *subsolutions_pool_h){
    
    hipSetDevice(gpu_id);
   // hipFuncSetCacheConfig(reinterpret_cast<const void*>(BP_queens_root_dfs),hipFuncCachePreferL1);
   
    unsigned long long *vector_of_tree_size_d;
    int *qtd_sols_d, *best_sol_d;
    Minla_node *subsolutions_pool_d;

    int num_blocks = ceil((double)pool_size/_MINLA_BLOCK_SIZE_);

    
    unsigned long long *vector_of_tree_size_h = (unsigned long long *) malloc (sizeof(unsigned long long)*pool_size);
    int *qtd_sols_h =  (int *) malloc (sizeof(int)*pool_size);
    int *best_sol_h =  (int *) malloc (sizeof(int)*pool_size);

    hipMalloc((void**) &vector_of_tree_size_d,pool_size*sizeof(unsigned long long));
    hipMalloc((void**) &qtd_sols_d,pool_size*sizeof(int));
    hipMalloc((void**) &best_sol_d, pool_size*sizeof(int));
    hipMalloc((void**) &subsolutions_pool_d,pool_size*sizeof(Minla_node));

    hipMemcpy(subsolutions_pool_d, subsolutions_pool_h, pool_size * sizeof(Minla_node), hipMemcpyHostToDevice);

    std::cout<<"### Regular BP-DFS search. ###\n";
    
    
    //BP_queens_root_dfs<<< num_blocks,_QUEENS_BLOCK_SIZE_>>> (size,n_explorers,initial_depth,root_prefixes_d, vector_of_tree_size_d,sols_d);
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipPeekAtLastError() );
    std::cout<<"### End of the GPU search ###\n";

    hipMemcpy(vector_of_tree_size_h,vector_of_tree_size_d,pool_size*sizeof(unsigned long long),hipMemcpyDeviceToHost);
    hipMemcpy(qtd_sols_h,qtd_sols_d,pool_size*sizeof(int),hipMemcpyDeviceToHost);

    for(int i = 0; i<pool_size;++i){
        local_tree_size+=vector_of_tree_size_h[i];
        *qtd_sols+=qtd_sols_h[i];
    }

    hipFree(vector_of_tree_size_d);
    hipFree(qtd_sols_d);
    hipFree(subsolutions_pool_d);
    //After that, Chapel reduces the values
}

__global__ void minla_gpu_node_explorer(int cutoff_depth, unsigned long long *tree_size, int *qtd_sols, int *best_sol,
    int** adj_list, Minla_node *pool, int upper_bound, int pool_size){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<pool_size){ //idx check

        unsigned flag = 0;
        unsigned bit_test = 0;
        int permutation[_MAX_]; //representa o ciclo
        int i, depth; //para dizer que 0-1 ja foi visitado e a busca comeca de 1, bote 2
     
        unsigned long long local_tree = 0ULL;
        int best_sol = upper_bound;
        
        int current_sol = 0;
        int num_sols = 0;
        int N = grafo->numNodes;
        int partial_cost = 0;
        int partial_sol = 0;    
        int stack[_MAX_];

        for (i = 0; i < N; ++i) { //
            permutation[i] = _EMPTY_;
        }

        depth = cutoff_depth;
        flag = pool[node_id].flag;
        partial_sol = pool[node_id].cost;

        for(i = 0; i<cutoff_depth;++i)
            permutation[i] = pool[node_id].permutation[i];
        
        while(true){ //search itself

            permutation[depth]++;
            bit_test = 0;
            bit_test |= (1<<permutation[depth]);

            if(permutation[depth] == N){ //all combinations for a given depth have been evaluated
                permutation[depth] = _EMPTY_;
            }
            else{

                if (!(flag & bit_test) ){ //is it valid?

                    partial_cost = grafo->ppartial_cost(permutation,depth+1);

                    if(partial_sol+partial_cost < best_sol){
                    
                        //cout<<" Leng: "<< depth+1<<" Partial cost: "<< current_sol<<" Cost test: "<<partial_sol+grafo->ppartial_cost(permutation,depth+1)<<"\n";
                        flag |= (1ULL<<permutation[depth]);
                        partial_sol += partial_cost; 
                        
                        stack[depth] = partial_cost;
                        
                        depth++;
                        ++local_tree;
                        
                        if (depth == N){ //a complete solution 
                            
                            ++num_sols;
                            best_sol = partial_sol;
                            //cout<<std::endl<<"Sol "<<num_sols<< " :"<<best_sol<<" "<<std::endl;
                            //for(int k = 1; k < N; k++){
                            //    std::cout   << " " << permutation[k];
                            //}

                        }//complete solution
                        else continue;
                    }//prune by value
                    else continue;
                }
                else continue;

            }//first else

            depth--; 
            
            //std::cout<<"\n alor";

            partial_sol-=stack[depth];
            flag &= ~(1ULL<<permutation[depth]);

            if(depth < cutoff_depth)
                break;
            //termination condition of the search

        }//end while -- end of the enumeration

        tree_size[idx] = local_tree;
        qtd_sols[idx]  = num_sols;
        best_sol[idx]  = best_sol;


    }//if idx   
 
}