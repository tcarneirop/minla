#include "hip/hip_runtime.h"
#include <limits>
#include <iostream>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <chrono>
#include "../headers/grafo.h"
#include "../headers/full_perm.h"
#include "../headers/minla_node.h"
#include "../headers/minla_kernels.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

#define _GPU_ADJ(v,j) gpu_adj_list[(v)*stride+(j)]

#define _CONST_GPU_ADJ(v,j) const_gpu_adj_list[(v)*const_stride+(j)]

__constant__ int const_gpu_adj_list[_MAX_*_MAX_];
__constant__ int const_gpu_size_adj_list[_MAX_];
__constant__ int const_N;
__constant__ int const_stride;


inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



__device__ inline int minla_gpu_Abs(int v){

    int const mask = v >> sizeof(int) * CHAR_BIT - 1;
    return (v + mask) ^ mask;
}

__device__ inline int minla_gpu_partial_cost(int *tag,int len, int *gpu_adj_list, int *gpu_size_adj_list, int stride){
    
    int sum = 0;
    int pos = len-1;
    int tag_pos = tag[pos];
    
    for(int j = 0 ; j < gpu_size_adj_list[pos]; j++){ //neighborhood of the vertex  

        if( _GPU_ADJ(pos,j) >= len) //not yet in the permutation
              continue;            
        
        sum += minla_gpu_Abs(tag_pos - tag[ _GPU_ADJ(pos,j) ]);
            
    }//for
    return sum;

}/////////////////////////////


__device__ inline int minla_const_gpu_partial_cost(int *tag,int len){
    
    int sum = 0;
    int pos = len-1;
    int tag_pos = tag[pos];
    
    for(int j = 0 ; j < const_gpu_size_adj_list[pos]; j++){ //neighborhood of the vertex  

        if( _CONST_GPU_ADJ(pos,j) >= len) //not yet in the permutation
              continue;            
        
        sum += minla_gpu_Abs(tag_pos - tag[ _CONST_GPU_ADJ(pos,j) ]);
            
    }//for
    return sum;

}/////////////////////////////


__global__ void minla_const_gpu_node_explorer(int cutoff_depth, unsigned long long *tree_size, int *qtd_sols, int *best_sols,  
    Minla_node *pool, int pool_size, int upper_bound){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(idx<pool_size){ //idx check

        unsigned flag = 0;
        unsigned bit_test = 0;
        int permutation[_MAX_]; //representa o ciclo
        int i, depth; //para dizer que 0-1 ja foi visitado e a busca comeca de 1, bote 2
     
        unsigned long long local_tree = 0ULL;
        int best_sol = upper_bound;
        
        int num_sols = 0;
        int N = const_N;
        int partial_cost = 0;
        int partial_sol = 0;    
        int stack[_MAX_];

        for (i = 0; i < N; ++i) { //
            permutation[i] = _EMPTY_;
        }

        depth = cutoff_depth;
        flag = pool[idx].flag;
        partial_sol = pool[idx].cost;

        for(i = 0; i<cutoff_depth;++i)
            permutation[i] = pool[idx].permutation[i];

        
        while(true){ //search itself

            permutation[depth]++;
            bit_test = 0;
            bit_test |= (1<<permutation[depth]);

            if(permutation[depth] == N){ //all combinations for a given depth have been evaluated
                permutation[depth] = _EMPTY_;
            }
            else{

                if (!(flag & bit_test) ){ //is it valid?

                    partial_cost = minla_const_gpu_partial_cost(permutation,depth+1);

                    if(partial_sol+partial_cost < best_sol){
                    
                        flag |= (1ULL<<permutation[depth]);
                        partial_sol += partial_cost; 
                        
                        stack[depth] = partial_cost;
                        
                        depth++;
                        ++local_tree;
                        
                        if (depth == N){ //a complete solution 
                            
                            ++num_sols;
                            best_sol = partial_sol;
                            num_sols+=1;

                        }//complete solution
                        else continue;
                    }//prune by value
                    else continue;
                }
                else continue;

            }//first else

            depth--; 
            
            partial_sol-=stack[depth];
            flag &= ~(1ULL<<permutation[depth]);

            if(depth < cutoff_depth)
                break;
            //termination condition of the search

        }//end while -- end of the enumeration

        tree_size[idx] = local_tree;
        qtd_sols[idx]  = num_sols;
        best_sols[idx] = best_sol;

    }//if idx   
 
}

__global__ void minla_gpu_node_explorer(int cutoff_depth, unsigned long long *tree_size, int *qtd_sols, int *best_sols,  
    int *gpu_adj_list, int *gpu_size_adj_list,
    Minla_node *pool, int pool_size, int upper_bound, int numNodes){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(idx<pool_size){ //idx check

        unsigned flag = 0;
        unsigned bit_test = 0;
        int permutation[_MAX_]; //representa o ciclo
        int i, depth; //para dizer que 0-1 ja foi visitado e a busca comeca de 1, bote 2
     
        unsigned long long local_tree = 0ULL;
        int best_sol = upper_bound;
        
        int num_sols = 0;
        int N = numNodes;
        int partial_cost = 0;
        int partial_sol = 0;    
        int stack[_MAX_];
        int stride = N-1;

        for (i = 0; i < N; ++i) { //
            permutation[i] = _EMPTY_;
        }

        depth = cutoff_depth;
        flag = pool[idx].flag;
        partial_sol = pool[idx].cost;

        for(i = 0; i<cutoff_depth;++i)
            permutation[i] = pool[idx].permutation[i];

        
        while(true){ //search itself

            permutation[depth]++;
            bit_test = 0;
            bit_test |= (1<<permutation[depth]);

            if(permutation[depth] == N){ //all combinations for a given depth have been evaluated
                permutation[depth] = _EMPTY_;
            }
            else{

                if (!(flag & bit_test) ){ //is it valid?

                    partial_cost = minla_gpu_partial_cost(permutation,depth+1, gpu_adj_list, gpu_size_adj_list, stride);

                    if(partial_sol+partial_cost < best_sol){
                    
                        flag |= (1ULL<<permutation[depth]);
                        partial_sol += partial_cost; 
                        
                        stack[depth] = partial_cost;
                        
                        depth++;
                        ++local_tree;
                        
                        if (depth == N){ //a complete solution 
                            
                            ++num_sols;
                            best_sol = partial_sol;
                            num_sols+=1;

                        }//complete solution
                        else continue;
                    }//prune by value
                    else continue;
                }
                else continue;

            }//first else

            depth--; 
            
            partial_sol-=stack[depth];
            flag &= ~(1ULL<<permutation[depth]);

            if(depth < cutoff_depth)
                break;
            //termination condition of the search

        }//end while -- end of the enumeration

        tree_size[idx] = local_tree;
        qtd_sols[idx]  = num_sols;
        best_sols[idx] = best_sol;

    }//if idx   
 
}


void minla_call_multigpu_kernel(int gpu_id, int cutoff_depth, unsigned long long *tree_size,
    int *qtd_sols, Grafo *grafo, int upper_bound, int pool_size,  Minla_node *subsolutions_pool_h){
    
    hipSetDevice(gpu_id);
   // hipFuncSetCacheConfig(reinterpret_cast<const void*>(BP_queens_root_dfs),hipFuncCachePreferL1);
   
    unsigned long long *vector_of_tree_size_d;
    int *qtd_sols_d, *best_sol_d,  *gpu_adj_list_d, *gpu_size_adj_list_d;
    
    Minla_node *subsolutions_pool_d;

    int local_qtd_sols = 0;
    unsigned long long local_tree_size = 0ULL;


    int num_blocks = ceil((double)pool_size/_MINLA_BLOCK_SIZE_);

    /////////////////////
    ////Host buffer
    ////////////////////
    int *qtd_sols_h =  (int *) malloc (sizeof(int)*pool_size);
    int *best_sol_h =  (int *) malloc (sizeof(int)*pool_size);
    unsigned long long *vector_of_tree_size_h = (unsigned long long *) malloc (sizeof(unsigned long long)*pool_size);


    /////////////////////
    ////GPU Buffer
    ////////////////////

    hipMalloc((void**) &vector_of_tree_size_d, pool_size*sizeof(unsigned long long));
    hipMalloc((void**) &qtd_sols_d,pool_size*sizeof(int));
    hipMalloc((void**) &best_sol_d, pool_size*sizeof(int));
    hipMalloc((void**) &gpu_adj_list_d, grafo->numNodes*(grafo->numNodes-1)*sizeof(int));
    hipMalloc((void**) &gpu_size_adj_list_d, grafo->numNodes*sizeof(int));
    hipMalloc((void**) &best_sol_d, pool_size*sizeof(int));
    hipMalloc((void**) &subsolutions_pool_d,pool_size*sizeof(Minla_node));

    //memcopy 

    hipMemcpy(subsolutions_pool_d, subsolutions_pool_h, pool_size * sizeof(Minla_node), hipMemcpyHostToDevice);
    hipMemcpy(gpu_adj_list_d,  grafo->gpu_adj_list, grafo->numNodes*(grafo->numNodes-1)*sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy(gpu_size_adj_list_d, grafo->gpu_size_adj_list, grafo->numNodes*sizeof(int), hipMemcpyHostToDevice);


    std::cout<<"### Regular BP-DFS search. ###\n";

    minla_gpu_node_explorer<<< num_blocks,_MINLA_BLOCK_SIZE_>>> (cutoff_depth, vector_of_tree_size_d, qtd_sols_d, best_sol_d,gpu_adj_list_d, 
        gpu_size_adj_list_d, subsolutions_pool_d, pool_size,upper_bound, grafo->numNodes);  

  

    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipPeekAtLastError() );
    std::cout<<"### End of the GPU search ###\n";

    hipMemcpy(vector_of_tree_size_h,vector_of_tree_size_d,pool_size*sizeof(unsigned long long),hipMemcpyDeviceToHost);
    hipMemcpy(qtd_sols_h,qtd_sols_d,pool_size*sizeof(int),hipMemcpyDeviceToHost);

    for(int i = 0; i<pool_size;++i){
        local_tree_size += vector_of_tree_size_h[i];
        local_qtd_sols  += qtd_sols_h[i];
    }

    std::cout<<"Local tree size: "<<local_tree_size<<"\n";
    std::cout<<"qtd_sols: "<<local_qtd_sols<<"\n";
    *tree_size += local_tree_size;
    *qtd_sols += local_qtd_sols;

    hipFree(vector_of_tree_size_d);
    hipFree(qtd_sols_d);
    hipFree(subsolutions_pool_d);
    //After that, Chapel reduces the values
}



void minla_call_const_multigpu_kernel(int gpu_id, int cutoff_depth, unsigned long long *tree_size,
    int *qtd_sols, Grafo *grafo, int upper_bound, int pool_size,  Minla_node *subsolutions_pool_h){
    
    hipSetDevice(gpu_id);
   // hipFuncSetCacheConfig(reinterpret_cast<const void*>(BP_queens_root_dfs),hipFuncCachePreferL1);
   
    unsigned long long *vector_of_tree_size_d;
    int *qtd_sols_d, *best_sol_d;
    
    Minla_node *subsolutions_pool_d;

    int local_qtd_sols = 0;
    unsigned long long local_tree_size = 0ULL;
    int stride = (grafo->numNodes)-1;


    int num_blocks = ceil((double)pool_size/_MINLA_BLOCK_SIZE_);

    /////////////////////
    ////Host buffer
    ////////////////////
    int *qtd_sols_h =  (int *) malloc (sizeof(int)*pool_size);
    int *best_sol_h =  (int *) malloc (sizeof(int)*pool_size);
    unsigned long long *vector_of_tree_size_h = (unsigned long long *) malloc (sizeof(unsigned long long)*pool_size);


    /////////////////////
    ////GPU Buffer
    ////////////////////

    hipMalloc((void**) &vector_of_tree_size_d, pool_size*sizeof(unsigned long long));
    hipMalloc((void**) &qtd_sols_d,pool_size*sizeof(int));
    hipMalloc((void**) &best_sol_d, pool_size*sizeof(int));
    hipMalloc((void**) &subsolutions_pool_d,pool_size*sizeof(Minla_node));

    //memcopy 

    hipMemcpy(subsolutions_pool_d, subsolutions_pool_h, pool_size * sizeof(Minla_node), hipMemcpyHostToDevice);
    //hipMemcpy(gpu_adj_list_d,  grafo->gpu_adj_list, grafo->numNodes*(grafo->numNodes-1)*sizeof(int) , hipMemcpyHostToDevice);
    //hipMemcpy(gpu_size_adj_list_d, grafo->gpu_size_adj_list, grafo->numNodes*sizeof(int), hipMemcpyHostToDevice);

    /// Copy to the constant region
    hipMemcpyToSymbol(HIP_SYMBOL(const_gpu_adj_list),  grafo->gpu_adj_list, grafo->numNodes*(grafo->numNodes-1)*sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(const_gpu_size_adj_list), grafo->gpu_size_adj_list, grafo->numNodes*sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(const_N), &grafo->numNodes, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(const_stride), &stride, sizeof(int));

    std::cout<<"### Const BP-DFS search. ###\n";

   // minla_gpu_node_explorer<<< num_blocks,_MINLA_BLOCK_SIZE_>>> (cutoff_depth, vector_of_tree_size_d, qtd_sols_d, best_sol_d,gpu_adj_list_d, 
   //     gpu_size_adj_list_d, subsolutions_pool_d, pool_size,upper_bound, grafo->numNodes);  


    minla_const_gpu_node_explorer<<< num_blocks,_MINLA_BLOCK_SIZE_>>> (cutoff_depth, vector_of_tree_size_d, qtd_sols_d, best_sol_d,subsolutions_pool_d, 
        pool_size,upper_bound);   
 

    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipPeekAtLastError() );
    std::cout<<"### End of the Const GPU search ###\n";

    hipMemcpy(vector_of_tree_size_h,vector_of_tree_size_d,pool_size*sizeof(unsigned long long),hipMemcpyDeviceToHost);
    hipMemcpy(qtd_sols_h,qtd_sols_d,pool_size*sizeof(int),hipMemcpyDeviceToHost);

    for(int i = 0; i<pool_size;++i){
        local_tree_size += vector_of_tree_size_h[i];
        local_qtd_sols  += qtd_sols_h[i];
    }

    std::cout<<"Local tree size: "<<local_tree_size<<"\n";
    std::cout<<"qtd_sols: "<<local_qtd_sols<<"\n";
    *tree_size += local_tree_size;
    *qtd_sols += local_qtd_sols;

    hipFree(vector_of_tree_size_d);
    hipFree(qtd_sols_d);
    hipFree(subsolutions_pool_d);
    //After that, Chapel reduces the values
}
