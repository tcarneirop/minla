#include "hip/hip_runtime.h"
#include <limits>
#include <iostream>
#include <omp.h>
#include <hip/hip_runtime.h
>#include <chrono>
#include "../headers/grafo.h"
#include "../headers/full_perm.h"
#include "../headers/minla_node.h"
#include "../headers/partial_search.h"
#include "../headers/minla_omp_search.h"
#include "../headers/minla_gpu_search.h"

#define _MINLA_BLOCK_SIZE_     128

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void minla_call_cuda_search(int cutoff_depth, Grafo *grafo, int upper_bound){

    int pool_size = 0; 
    int qtd_sol = 0;
    unsigned long long initial_search_tree_size = 0ULL;
    unsigned long long final_search_tree_size = 0ULL;
    int best_sol = upper_bound;
    std::cout.precision(5);

    std::cout <<"\n Partial search -  Cutoff depth: " << cutoff_depth<<"\n";

    std::chrono::steady_clock clk;   // create an object of `steady_clock` class
    
    auto start = clk.now();


    Minla_node *subsolutions_pool = minla_start_pool(grafo, cutoff_depth);
    minla_partial_search(cutoff_depth, &initial_search_tree_size, &qtd_sol, grafo, subsolutions_pool, upper_bound);
    pool_size = qtd_sol;
   // minla_print_pool(subsolutions_pool, pool_size, cutoff_depth);
    
    std::cout<<"Maximum pool size: "<<minla_max_pool_size(grafo,cutoff_depth)<<"\n";
    std::cout<<std::endl<<std::endl<<"Pool size: "<<pool_size<<"\n";

    qtd_sol = 0;
    
    auto end = clk.now();       // end timer (starting & ending is done by measuring the time at the moment the process started & ended respectively)
    auto time_span = static_cast<std::chrono::duration<double>>(end - start);   // measure time span between start & end

    unsigned long long total_tree = initial_search_tree_size+final_search_tree_size;
    std::cout<<"Number of solutions found: "<<qtd_sol<<"\n\tOptimal solution: "<<upper_bound<<"\n\n";
    std::cout<<"Tree size: "<<total_tree<<"\n";
    std::cout<<fixed<<"Performance: "<<total_tree/time_span.count()<< " nodes/sec\n";
    std::cout<<"\nElapsed time: "<< time_span.count() <<" seconds"<<"\n";

}


int GPU_minla_call_multigpu_kernel(int gpu_id, int cutoff_depth, unsigned long long *local_tree_size, int *qtd_sol, Grafo *grafo, int upper_bound, int pool_size,  Minla_node *subsolutions_pool_h){
    
    hipSetDevice(gpu_id);
   // hipFuncSetCacheConfig(reinterpret_cast<const void*>(BP_queens_root_dfs),hipFuncCachePreferL1);
   

    unsigned long long *vector_of_tree_size_d;
    int *qtd_sols_d;
    Minla_node *subsolutions_pool_d;

    int num_blocks = ceil((double)pool_size/_MINLA_BLOCK_SIZE_);

    hipMalloc((void**) &vector_of_tree_size_d,pool_size*sizeof(unsigned long long));
    hipMalloc((void**) &qtd_sols_d,pool_size*sizeof(int));
    hipMalloc((void**) &subsolutions_pool_d,pool_size*sizeof(Minla_node));


    hipMemcpy(subsolutions_pool_d, subsolutions_pool_h, pool_size * sizeof(Minla_node), hipMemcpyHostToDevice);

    std::cout<<"### Regular BP-DFS search. ###\n";
    
    
    BP_queens_root_dfs<<< num_blocks,_QUEENS_BLOCK_SIZE_>>> (size,n_explorers,initial_depth,root_prefixes_d, vector_of_tree_size_d,sols_d);
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipPeekAtLastError() );
  

    hipMemcpy(vector_of_tree_size_h,vector_of_tree_size_d,pool_size*sizeof(unsigned long long),hipMemcpyDeviceToHost);
    hipMemcpy(qtd_sols_h,qtd_sols_d,pool_size*sizeof(int),hipMemcpyDeviceToHost);

    for(int i = 0; i<pool_size;++i){
        local_tree_size+=vector_of_tree_size_h[i];
        qtd_sols+=qtd_sols_h[i];
    }

    hipFree(vector_of_tree_size_d);
    hipFree(qtd_sols_d);
    hipFree(subsolutions_pool_d);
    //After that, Chapel reduces the values
}
